#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include <stdio.h>

__global__ void mergeKernel(
  double* z_real_dev, 
  double* z_real_new_dev, 
  double* z_imag_dev, 
  double* z_imag_new_dev
) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ double z_real_shr[512];
  __shared__ double z_imag_shr[512];

  z_real_shr[threadIdx.x] = z_real_dev[tid];
  z_imag_shr[threadIdx.x] = z_imag_dev[tid];
  __syncthreads();

  z_real_shr[threadIdx.x] += z_real_new_dev[tid];
  z_imag_shr[threadIdx.x] += z_imag_new_dev[tid];
  __syncthreads();

  z_real_dev[tid] = z_real_shr[threadIdx.x];
  z_imag_dev[tid] = z_imag_shr[threadIdx.x];
}

void check(hipError_t err, const char *const func, const char *const file,
           const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    // We don't exit when we encounter CUDA errors in this example.
    // std::exit(EXIT_FAILURE);
  }
}


void call_kernel(
  size_t nDim,
  double* z_real_dev, 
  double* z_real_new_dev, 
  double* z_imag_dev, 
  double* z_imag_new_dev
) {
  mergeKernel<<<nDim / 512, 512>>>(z_real_dev, z_real_new_dev,
    z_imag_dev, z_imag_new_dev
  );

  hipDeviceSynchronize();

  hipError_t err = hipGetLastError();

  CHECK_CUDA_ERROR(err);
}